#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

#define threads_per_block 2
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void Q1(float* a,float *b,float *c,size_t N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < N){
        c[i] = a[i] + b[i];
    }
}

int main(){

    size_t N = 1000000;

    float *a = new float[N];
    float *b = new float[N];
    float *c = new float[N];

    dim3 threads(threads_per_block,1,1);
    dim3 blocks((N + threads_per_block - 1) / threads_per_block,1,1);

    float *a_d,*b_d,*c_d;
    cudaCheck(hipMalloc(&a_d, N * sizeof(float)));
    cudaCheck(hipMalloc(&b_d, N * sizeof(float)));
    cudaCheck(hipMalloc(&c_d, N * sizeof(float)));

    cudaCheck(hipMemcpy(a_d, a, N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(b_d, b, N * sizeof(float), hipMemcpyHostToDevice));


    Q1<<<blocks,threads>>>(a_d,b_d,c_d,N);
    cudaCheck(hipGetLastError());
    cudaCheck(hipDeviceSynchronize());

    cudaCheck(hipMemcpy(c, c_d, N * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    delete[] a;
    delete[] b;
    delete[] c;
}