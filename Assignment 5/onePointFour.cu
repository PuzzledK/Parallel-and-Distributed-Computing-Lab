#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

#define threads_per_block 256
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
const size_t N = 100000000;


inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void Q1(float* a,float* b,float* c,size_t N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < N){
        c[i] = a[i] + b[i];
    }
}

int main(){

    hipEvent_t start,stop;
    float ms = 0;

    float *a = new float[N];
    float *b = new float[N];
    float *c = new float[N];

    dim3 threads(threads_per_block,1,1);
    dim3 blocks((N + threads_per_block - 1) / threads_per_block,1,1);

    float *a_d,*b_d,*c_d;
    cudaCheck(hipMalloc(&a_d, N * sizeof(float)));
    cudaCheck(hipMalloc(&b_d, N * sizeof(float)));
    cudaCheck(hipMalloc(&c_d, N * sizeof(float)));

    cudaCheck(hipMemcpy(a_d, a, N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(b_d, b, N * sizeof(float), hipMemcpyHostToDevice));

    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));

    cudaCheck(hipEventRecord(start));

    Q1<<<blocks,threads>>>(a_d,b_d,c_d,N);
    cudaCheck(hipGetLastError());

    cudaCheck(hipEventRecord(stop));
    cudaCheck(hipEventSynchronize(stop));

    cudaCheck(hipEventElapsedTime(&ms,start,stop));
    cudaCheck(hipMemcpy(c, c_d, N * sizeof(float), hipMemcpyDeviceToHost));


    float bytes = (N * (2 * sizeof(float) + sizeof(float))) / (1e9f);

    cout<<"KERNEL EXECUTION TIME -> "<<ms<<" milliseconds"<<endl;

    cout<<"BANDWIDTH -> "<< bytes / (ms/1000.0f)<<" GB/s"<<endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    delete[] a;
    delete[] b;
    delete[] c;
}